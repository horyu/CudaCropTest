#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "imageIO.h"
#include "cudaCrop.h"

int main()
{
    const char* input_path = R"(C:\Users\owner\source\repos\CudaCropTest\CudaCropTest\images\sample-2000x1500.png)";
    const char* output_oath = R"(C:\Users\owner\source\repos\CudaCropTest\CudaCropTest\images\test.png)";

    uchar3 *input_image = NULL;
    int width = 0, height = 0;

    // Load image
    if (!loadImage(input_path, &input_image, &width, &height)) {
        fprintf(stderr, "loadImage failed!");
		return 1;
    }
    printf("Image loaded: %d x %d\n", width, height);

    // Save image
    if (!saveImage(output_oath, input_image, width, height)) {
		fprintf(stderr, "saveImage failed!");
        return 1;
    }

    return 0;
}
