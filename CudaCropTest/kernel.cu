#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "imageIO.h"
#include "cudaCrop.h"
#include "cudaMappedMemory.h"

class CudaMemory {
public:
	CudaMemory(uchar3* ptr) : ptr_(ptr) {}
	~CudaMemory() { hipFree(ptr_); }
private:
	uchar3* ptr_;
};

int main()
{
	const char* input_path = R"(C:\Users\owner\source\repos\CudaCropTest\CudaCropTest\images\sample-2000x1500.png)";
	const char* output_path = R"(C:\Users\owner\source\repos\CudaCropTest\CudaCropTest\images\test.png)";

	uchar3* input_image = NULL;
	uchar3* output_image = NULL;
	CudaMemory input_memory(input_image), output_memory(output_image);

	int width = 0, height = 0;

	// Load image
	if (!loadImage(input_path, &input_image, &width, &height)) {
		fprintf(stderr, "loadImage failed!");
		return 1;
	}
	printf("Image loaded: %d x %d\n", width, height);

	int crop_width = width / 2;
	int crop_height = height / 3;

	// Allocate output image
	if (!cudaAllocMapped(&output_image, sizeof(uchar3) * crop_width * crop_height)) {
		fprintf(stderr, "cudaAllocMapped failed!");
		return 1;
	}

	// Crop image
	int4 roi = {
		width / 4,
		height / 3,
		width / 4 + crop_width,
		height / 3 + crop_height
	};
	hipError_t cudaStatus = cudaCrop(
		input_image,
		output_image,
		roi,
		width,
		height
	);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCrop failed! cudaStatus=%d", cudaStatus);
		return 1;
	}

	// Save image
	if (!saveImage(output_path, output_image, crop_width, crop_height)) {
		fprintf(stderr, "saveImage failed!");
		return 1;
	}

	return 0;
}
